
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 512
//conv filter used here is identity
__global__ void convBaseline(double *in, double *out, int vector_size, int RADIUS) {
	// each thread computes result for 1 element of output conv
    int gindex = threadIdx.x + blockIdx.x * blockDim.x; //global index
	if (gindex < vector_size){
		double result = 0.0;
		for (int offset = -RADIUS ; offset <= RADIUS ; ++offset)
			result += (gindex + offset >= 0 && gindex + offset < vector_size) ? in[gindex + offset] : 0.0;
		out[gindex] = result; 
	}
}	

int main( int argc, char* argv[] ) { 
	if (argc != 3) {
		printf ("Usage: %s vector_size radiusSize\n", argv[0]);
		return 1;
	}
	
	int vector_size = atoi(argv[1]);
	int RADIUS = atoi(argv[2]);
	int grid_size   = ((vector_size-1)/BLOCK_SIZE) + 1;

	hipSetDevice(0);
        
	hipEvent_t start, stop;
	float time;
	hipEventCreate (&start);
	hipEventCreate (&stop);

	
	double *in_cpu         = new double [vector_size]; // CPU Struct
	double *out_cpu        = new double [vector_size];
	double *out_gpu_on_cpu = new double [vector_size]; // Copy GPU result to CPU

	printf("Initializing input arrays.\n");
	for (int i = 0; i < vector_size; i++) { // fill the arrays 'a' and 'b' on the CPU
		in_cpu[i] = (rand()%100)*cos(i);
		out_cpu[i] = 0.0;
		out_gpu_on_cpu[i] = 0.0;
	}
	
	double *in_gpu; // GPU pointers
	double *out_gpu;
	hipMalloc (&in_gpu, vector_size*sizeof(double));
	hipMalloc (&out_gpu, vector_size*sizeof(double));

	hipMemcpy (in_gpu, in_cpu, vector_size*sizeof(double), hipMemcpyHostToDevice);

	printf("Running sequential job.\n");
	hipEventRecord(start,0);

	for (int i = 0; i < vector_size; ++i) { // Calculate C in the CPU
		for (int offset = -RADIUS ; offset <= RADIUS ; ++offset)
		   out_cpu[i] += (i + offset >= 0 && i + offset < vector_size) ? in_cpu[i + offset] : 0.0;
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("\tSequential Job Time: %.2f ms\n", time);

	printf("Running parallel job.\n");

	hipEventRecord(start,0);

	convBaseline<<<grid_size, BLOCK_SIZE>>>(in_gpu, out_gpu, vector_size,RADIUS);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);
	printf("\tParallel Job Time: %.2f ms\n", time);

	hipMemcpy (out_gpu_on_cpu, out_gpu, vector_size*sizeof(double), hipMemcpyDeviceToHost);
	
	int error = 0;
	for (int i = 0; i < vector_size; i++) {
		if (out_cpu[i] != out_gpu_on_cpu[i]){
			error = 1;
			printf( "Mistake at element %d\n", i);
			int start = (i-RADIUS<0)?0:i-RADIUS;
			int end = (i+RADIUS>vector_size)?vector_size:i+RADIUS;
			for (int offset = start ; offset <= end ; offset++)
                printf( "index = %d \tin = %.5lf \tout GPU = %.5lf \tCPU %.5lf\n", offset, 
															in_cpu[offset], 
															out_gpu_on_cpu[offset], 
															out_cpu[offset] );    
		}
		if (error) break; 
	}

	if (error == 0){
		printf ("Correct result. No errors were found.\n");
	}

	free (in_cpu); //free GPU
	free (out_cpu);
	free (out_gpu_on_cpu);
	hipFree (in_gpu); //free GPU
	hipFree (out_gpu);

	return 0;
}
