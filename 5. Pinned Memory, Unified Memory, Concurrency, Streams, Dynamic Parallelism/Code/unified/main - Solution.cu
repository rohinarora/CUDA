#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void initialize (int N, float *a, float *b, float *c){
	int i = threadIdx.x+blockIdx.x*blockDim.x;
	if (i < N){
		c[i] = 0;
		a[i] = 1 + i;
		b[i] = 1 - i;
	}
}

__global__ void addVectors (int N, float *a, float *b, float *c){
	int i = threadIdx.x+blockIdx.x*blockDim.x;
	if (i < N){
		c[i] = a[i] + b[i];
	}
}

int main (int argc, char **argv){
	
	if (argc != 2) exit (1);
	int N = atoi(argv[1]);

	float *a, *b, *c;
	hipMallocManaged (&a, N*sizeof(float));
	hipMallocManaged (&b, N*sizeof(float));
	hipMallocManaged (&c, N*sizeof(float));

	dim3 block(1024);
	dim3 grid((N-1)/1024+1);

	initialize<<<grid, block>>>(N,a,b,c);
	
	addVectors<<<grid, block>>>(N,a,b,c);

	hipDeviceSynchronize ();

	for (int i = 0; i < 5; i++) {
		printf("%f\n", c[i]);
	}

	hipFree(a);
	hipFree(b);
	hipFree(c);
}
