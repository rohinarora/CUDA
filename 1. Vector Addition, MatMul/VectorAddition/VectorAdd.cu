
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add( int *a, int *b, int *c, int vector_size ) { // must be pointers in GPU memory
    int tid = blockDim.x*blockIdx.x+threadIdx.x; //which thread is running
    if (tid < vector_size){   // Don't want threads with tid > vector_size to run this code. tid > vector_size -> this condition will be true for certain threads in last block.
      c[tid] = a[tid]+b[tid];
    }
}

int main( int argc, char* argv[] ) { 
    if (argc != 3) {
        printf ("Usage: %s vector_size block_size\n", argv[0]);
        return 1;
    }
    int vector_size = atoi(argv[1]);
    int block_size  = atoi(argv[2]);
    int grid_size   = ((vector_size-1)/block_size) + 1;

    hipSetDevice(0); // Set device that we will use for our cuda code
    hipEvent_t start, stop; // Time Variables
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    int *a        = new int [vector_size];  // Input Arrays and variables
    int *b        = new int [vector_size]; 
    int *c_cpu    = new int [vector_size]; 
    int *c_gpu    = new int [vector_size];

    printf("Initializing input arrays.\n"); // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < vector_size; i++) {
        a[i] = rand()%10;
        b[i] = rand()%10;
    }

    printf("Running sequential job.\n"); // CPU Calculation
    hipEventRecord(start,0);
    for (int i = 0; i < vector_size; i++) { // Calculate C in the CPU
            c_cpu[i] = a[i] + b[i];
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("\tSequential Job Time: %.2f ms\n", time);

    int *dev_a; // Pointers in GPU memory
    int *dev_b;
    int *dev_c;

    hipMalloc((void **)&dev_a, sizeof(int)*vector_size); // allocate the memory on the GPU
    hipMalloc((void **)&dev_b, sizeof(int)*vector_size);
    hipMalloc((void **)&dev_c, sizeof(int)*vector_size);

    hipMemcpy(dev_a,a,sizeof(float)*vector_size,hipMemcpyHostToDevice); // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_b,b,sizeof(float)*vector_size,hipMemcpyHostToDevice);

    printf("Running parallel job.\n");
    hipEventRecord(start,0);
    add<<<grid_size,block_size>>>(dev_a,dev_b,dev_c,vector_size); // GPU Calculation
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("\tParallel Job Time: %.2f ms\n", time);

    hipMemcpy(c_gpu,dev_c,sizeof(float)*vector_size,hipMemcpyDeviceToHost); // copy the array 'c' back from the GPU to the CPU
    
    int error = 0; // compare the results
    for (int i = 0; i < vector_size; i++) {
        if (c_cpu[i] != c_gpu[i]){
            error = 1;
            printf( "Error starting element %d, %d != %d\n", i, c_gpu[i], c_cpu[i] );    
        }
        if (error) break; 
    }

    if (error == 0){
        printf ("Correct result. No errors were found.\n");
    }

    free (a); // free CPU data
    free (b);
    free (c_cpu);
    free (c_gpu);
    hipFree(dev_a); // free the memory allocated on the GPU
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}