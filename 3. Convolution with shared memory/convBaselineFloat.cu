
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define RADIUS 8
#define BLOCK_SIZE 512
//conv filter used here is identity
__global__ void convBaseline(float *in, float *out, int vector_size) {
	// each thread computes result for 1 element of output conv
    int gindex = threadIdx.x + blockIdx.x * blockDim.x; //global index
	if (gindex < vector_size){
		float result = 0.0;
		for (int offset = -RADIUS ; offset <= RADIUS ; ++offset)
			result += (gindex + offset >= 0 && gindex + offset < vector_size) ? in[gindex + offset] : 0.0;
		out[gindex] = result; 
	}
}	

int main( int argc, char* argv[] ) { 
	if (argc != 2) {
		printf ("Usage: %s vector_size\n", argv[0]);
		return 1;
	}
	
	int vector_size = atoi(argv[1]);
	int grid_size   = ((vector_size-1)/BLOCK_SIZE) + 1;

	hipSetDevice(0);
        
	hipEvent_t start, stop;
	float time;
	hipEventCreate (&start);
	hipEventCreate (&stop);

	
	float *in_cpu         = new float [vector_size]; // CPU Struct
	float *out_cpu        = new float [vector_size];
	float *out_gpu_on_cpu = new float [vector_size]; // Copy GPU result to CPU

	printf("Initializing input arrays.\n");
	for (int i = 0; i < vector_size; i++) { // fill the arrays 'a' and 'b' on the CPU
		in_cpu[i] = (rand()%100)*cos(i);
		out_cpu[i] = 0.0;
		out_gpu_on_cpu[i] = 0.0;
	}
	
	float *in_gpu; // GPU pointers
	float *out_gpu;
	hipMalloc (&in_gpu, vector_size*sizeof(float));
	hipMalloc (&out_gpu, vector_size*sizeof(float));

	hipMemcpy (in_gpu, in_cpu, vector_size*sizeof(float), hipMemcpyHostToDevice);

	printf("Running sequential job.\n");
	hipEventRecord(start,0);

	for (int i = 0; i < vector_size; ++i) { // Calculate C in the CPU
		for (int offset = -RADIUS ; offset <= RADIUS ; ++offset)
		   out_cpu[i] += (i + offset >= 0 && i + offset < vector_size) ? in_cpu[i + offset] : 0.0;
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("\tSequential Job Time: %.2f ms\n", time);

	printf("Running parallel job.\n");

	hipEventRecord(start,0);

	convBaseline<<<grid_size, BLOCK_SIZE>>>(in_gpu, out_gpu, vector_size);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);
	printf("\tParallel Job Time: %.2f ms\n", time);

	hipMemcpy (out_gpu_on_cpu, out_gpu, vector_size*sizeof(float), hipMemcpyDeviceToHost);
	
	int error = 0;
	for (int i = 0; i < vector_size; i++) {
		if (out_cpu[i] != out_gpu_on_cpu[i]){
			error = 1;
			printf( "Mistake at element %d\n", i);
			int start = (i-RADIUS<0)?0:i-RADIUS;
			int end = (i+RADIUS>vector_size)?vector_size:i+RADIUS;
			for (int offset = start ; offset <= end ; offset++)
                printf( "index = %d \tin = %.5lf \tout GPU = %.5lf \tCPU %.5lf\n", offset, 
															in_cpu[offset], 
															out_gpu_on_cpu[offset], 
															out_cpu[offset] );    
		}
		if (error) break; 
	}

	if (error == 0){
		printf ("Correct result. No errors were found.\n");
	}

	free (in_cpu); //free GPU
	free (out_cpu);
	free (out_gpu_on_cpu);
	hipFree (in_gpu); //free GPU
	hipFree (out_gpu);

	return 0;
}
