
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE   512

typedef struct Data {
    
	double *a;
	double *b;
	double *c;
	
} Data;

__global__ void add( Data data, int vector_size ) {
	
	// Calculate the index in the vector for the thread using the internal variables
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	
	// This if statement is added in case we have more threads executing
	// Than number of elements in the vectors. How can this help?
	if (tid < vector_size){
		
		// Compute the addition
		data.c[tid] = data.a[tid] + data.b[tid];
		
	}
}

int main( int argc, char* argv[] ) { 

	// Parse Input arguments

	// Check the number of arguments (we only receive command + vector size)
	if (argc != 2) {
		// Tell the user how to run the program
		printf ("Usage: %s vector_size\n", argv[0]);
		// "Usage messages" are a conventional way of telling the user
		// how to run a program if they enter the command incorrectly.
		return 1;
	}
	
	// Set GPU Variables based on input arguments
	int vector_size = atoi(argv[1]);
	int grid_size   = ((vector_size-1)/BLOCK_SIZE) + 1;

	// Set device that we will use for our cuda code
	// It will be 0, 1, 2 or 3
	hipSetDevice(0);
        
	// Time Variables
	hipEvent_t start, stop;
	float time;
	hipEventCreate (&start);
	hipEventCreate (&stop);

	// CPU Struct
	Data data_cpu;
    // HERE
	//data_cpu.a = new double [vector_size]; 
	//data_cpu.b = new double [vector_size]; 
	data_cpu.c = new double [vector_size]; 
	Data data_gpu_on_cpu;
	//data_gpu_on_cpu.c = new double [vector_size]; 
            
    hipHostMalloc((void**)&data_cpu.a, vector_size*sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&data_cpu.b, vector_size*sizeof(double), hipHostMallocDefault);
    hipHostMalloc((void**)&data_gpu_on_cpu.c, vector_size*sizeof(double), hipHostMallocDefault);

	// fill the arrays 'a' and 'b' on the CPU
	printf("Initializing input arrays.\n");
	for (int i = 0; i < vector_size; i++) {
		data_cpu.a[i] = rand()*cos(i);
		data_cpu.b[i] = rand()*sin(i);
		data_cpu.c[i] = 0.0;
	}
	
	// allocate the memory on the GPU
	Data data_gpu; 
	hipMalloc (&data_gpu.a, vector_size*sizeof(double));
	hipMalloc (&data_gpu.b, vector_size*sizeof(double));
	hipMalloc (&data_gpu.c, vector_size*sizeof(double));
	
	//
	// CPU Calculation
	//////////////////

	printf("Running sequential job.\n");
	hipEventRecord(start,0);

	// Calculate C in the CPU
	for (int i = 0; i < vector_size; i++) {
			data_cpu.c[i] = data_cpu.a[i] + data_cpu.b[i];
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("\tSequential Job Time: %.2f ms\n", time);

	//
	// GPU Calculation
	////////////////////////

	printf("Running parallel job.\n");

	hipEventRecord(start,0);

	// copy the input to the GPU
	hipMemcpy (data_gpu.a, data_cpu.a, vector_size*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy (data_gpu.b, data_cpu.b, vector_size*sizeof(double), hipMemcpyHostToDevice);

	// call the kernel
	add<<<grid_size, BLOCK_SIZE>>>(data_gpu, vector_size);

	// copy the array 'c' back from the GPU to the CPU
	hipMemcpy (data_gpu_on_cpu.c, data_gpu.c, vector_size*sizeof(double), hipMemcpyDeviceToHost);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);
	printf("\tParallel Job Time: %.2f ms\n", time);
	
	// compare the results
	int error = 0;
	for (int i = 0; i < vector_size; i++) {
		if (data_cpu.c[i] != data_gpu_on_cpu.c[i]){
			error = 1;
			printf( "Error starting element %d, %f != %f\n", i, data_gpu_on_cpu.c[i], data_cpu.c[i] );    
		}
		if (error) break; 
	}

	if (error == 0){
		printf ("Correct result. No errors were found.\n");
	}

	// free CPU data
    // HERE
	hipHostFree (data_cpu.a);
	hipHostFree (data_cpu.b);
	free (data_cpu.c);
	hipHostFree (data_gpu_on_cpu.c);

	// free the memory allocated on the GPU
	hipFree (data_gpu.a);
	hipFree (data_gpu.b);
	hipFree (data_gpu.c);

	return 0;
}