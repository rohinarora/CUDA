
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE   512

typedef struct Data {
	double a;
	double b;
	double c;
	
} Data;

__global__ void add( Data *data, int vector_size ) {
	
	// Calculate the index in the vector for the thread using the internal variables
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	
	// This if statement is added in case we have more threads executing
	// Than number of elements in the vectors. How can this help?
	if (tid < vector_size){
		
		// Compute the addition
		data[tid].c = data[tid].a + data[tid].b;
		
	}
}

int main( int argc, char* argv[] ) { 

	// Parse Input arguments

	// Check the number of arguments (we only receive command + vector size)
	if (argc != 2) {
		// Tell the user how to run the program
		printf ("Usage: %s vector_size\n", argv[0]);
		// "Usage messages" are a conventional way of telling the user
		// how to run a program if they enter the command incorrectly.
		return 1;
	}
	
	// Set GPU Variables based on input arguments
	int vector_size = atoi(argv[1]);
	int grid_size   = ((vector_size-1)/BLOCK_SIZE) + 1;

	// Set device that we will use for our cuda code
	// It will be 0, 1, 2 or 3
	hipSetDevice(0);
        
	// Time Variables
	hipEvent_t start, stop;
	float time;
	hipEventCreate (&start);
	hipEventCreate (&stop);

	// CPU Struct
	Data *data_cpu        = new Data [vector_size]; 
	Data *data_gpu_on_cpu = new Data [vector_size]; 

	// fill the arrays 'a' and 'b' on the CPU
	printf("Initializing input arrays.\n");
	for (int i = 0; i < vector_size; i++) {
		data_cpu[i].a = rand()*cos(i);
		data_cpu[i].b = rand()*sin(i);
		data_cpu[i].c = 0.0;
	}
	
	// allocate the memory on the GPU
	Data *data_gpu; 
	hipMalloc (&data_gpu, vector_size*sizeof(Data));

	// copy the input to the GPU
	hipMemcpy (data_gpu, data_cpu, vector_size*sizeof(Data), hipMemcpyHostToDevice);
	
	//
	// CPU Calculation
	//////////////////

	printf("Running sequential job.\n");
	hipEventRecord(start,0);

	// Calculate C in the CPU
	for (int i = 0; i < vector_size; i++) {
			data_cpu[i].c = data_cpu[i].a + data_cpu[i].b;
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("\tSequential Job Time: %.2f ms\n", time);

	//
	// GPU Calculation
	////////////////////////

	printf("Running parallel job.\n");

	hipEventRecord(start,0);

	// call the kernel
	add<<<grid_size, BLOCK_SIZE>>>(data_gpu, vector_size);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);
	printf("\tParallel Job Time: %.2f ms\n", time);

	// copy the array 'c' back from the GPU to the CPU
	hipMemcpy (data_gpu_on_cpu, data_gpu, vector_size*sizeof(Data), hipMemcpyDeviceToHost);
	
	// compare the results
	int error = 0;
	for (int i = 0; i < vector_size; i++) {
		if (data_cpu[i].c != data_gpu_on_cpu[i].c){
			error = 1;
			printf( "Error starting element %d, %f != %f\n", i, data_gpu_on_cpu[i].c, data_cpu[i].c );    
		}
		if (error) break; 
	}

	if (error == 0){
		printf ("Correct result. No errors were found.\n");
	}

	// free CPU data
	free (data_cpu);
	free (data_gpu_on_cpu);

	// free the memory allocated on the GPU
	hipFree (data_gpu);

	return 0;
}

